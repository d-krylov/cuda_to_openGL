#include "hip/hip_runtime.h"
#include "renderer.h"

__global__ void Draw(float *image, uint32_t w, uint32_t h) {

  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  auto y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > w || y > h)
    return;

  int vertex_index = y * w + x;

  float u = float(x) / float(w);
  float v = float(y) / float(h);

  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;

  image[5 * vertex_index + 0] = u;
  image[5 * vertex_index + 1] = v;
  image[5 * vertex_index + 2] = std::sqrt(u * u + v * v);
  image[5 * vertex_index + 3] = 0.0f;
  image[5 * vertex_index + 4] = 0.0f;
}

void RunKernel(float *data, uint32_t w, uint32_t h) {
  dim3 threads(32, 32);

  int bx = w / threads.x;
  int by = h / threads.y;

  dim3 blocks(bx, by);

  Draw<<<blocks, threads>>>(data, w, h);
}
