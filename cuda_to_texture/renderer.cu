#include "hip/hip_runtime.h"
#include "common.h"
#include "renderer.h"

__global__ void Draw(uint32_t *image, uint32_t w, uint32_t h) {

  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  auto y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x > w || y > h)
    return;

  int vertex_index = y * w + x;

  image[vertex_index] = 0xff0000ff;
}

Renderer::Renderer(uint32_t w, uint32_t h) : width_(w), height_(h) {
  hipMalloc(&data_, w * h * sizeof(uint32_t));
}

Renderer::~Renderer() { hipFree(data_); }

void Renderer::RunKernel(hipArray *texture) {
  dim3 threads(32, 32);

  int bx = width_ / threads.x;
  int by = height_ / threads.y;

  dim3 blocks(bx, by);

  Draw<<<blocks, threads>>>(data_, width_, height_);

  auto pitch = width_ * sizeof(uint32_t);

  CUDA_CHECK(hipMemcpy2DToArray(texture, 0, 0, data_, pitch, pitch, height_,
                                 hipMemcpyDeviceToDevice));
}
