#include "common.h"
#include "cuda_graphics_resource.h"
#include <iostream>

CUDAGraphicsResource::CUDAGraphicsResource() {
  uint32_t count;
  int32_t id[16];

  auto status = hipGLGetDevices(&count, id, 1, hipGLDeviceListAll);

  if (status != hipSuccess) {
    std::cout << "First error" << std::endl;
  }

  status = hipSetDevice(id[0]);

  if (status != hipSuccess) {
  }
}

void CUDAGraphicsResource::RegisterBuffer(const Buffer &buffer) {
  CUDA_CHECK(hipGraphicsGLRegisterBuffer(&cuda_resource_, buffer.GetID(),
                                          cudaGraphicsMapFlagsNone));
}

void CUDAGraphicsResource::RegisterTexture(const Texture &texture) {
  CUDA_CHECK(hipGraphicsGLRegisterImage(&cuda_resource_, texture.GetID(),
                                         GL_TEXTURE_2D,
                                         cudaGraphicsMapFlagsWriteDiscard));
}

MappedPointer CUDAGraphicsResource::GetMappedPointer() {
  void *device_pointer;
  uint64_t size;
  CUDA_CHECK(hipGraphicsResourceGetMappedPointer(&device_pointer, &size,
                                                  cuda_resource_));
  return MappedPointer{device_pointer, size};
}

hipArray *CUDAGraphicsResource::GetMappedArray() {
  hipArray *array;
  CUDA_CHECK(
      hipGraphicsSubResourceGetMappedArray(&array, cuda_resource_, 0, 0));
  return array;
}

void CUDAGraphicsResource::Begin() {
  CUDA_CHECK(hipGraphicsMapResources(1, &cuda_resource_, 0));
}

void CUDAGraphicsResource::End() {
  CUDA_CHECK(hipGraphicsUnmapResources(1, &cuda_resource_, 0));
}
